

__host__            hipError_t hipEventCreate      ( hipEvent_t* event );
__host__ __device__ hipError_t hipEventDestroy     ( hipEvent_t  event );
__host__            hipError_t hipEventElapsedTime ( float* ms, hipEvent_t start, hipEvent_t end );
__host__            hipError_t hipEventQuery       ( hipEvent_t event );
__host__ __device__ hipError_t hipEventRecord      ( hipEvent_t event, hipStream_t stream = 0) ;
__host__            hipError_t hipEventSynchronize ( hipEvent_t event);
__host__            hipError_t hipStreamCreate     ( hipStream_t*stream );
__host__ __device__ hipError_t hipStreamDestroy    ( hipStream_t stream );
__host__ __device__ hipError_t hipStreamWaitEvent  ( hipStream_t stream, hipEvent_t event, unsigned int  flags = 0 );


int f()
{
  int numBlocks, numThreads, numSharedBytes, size;
  
  int *hBuff1, *hBuff2, *dBuff1, *dBuff2;

  float time;
    
  hipStream_t transferStream, workStream;

  hipStreamCreate(&transferStream);
  hipStreamCreate(&workStream);
    
  hipEvent_t empty1, empty2, full1, full2;

  hipEventCreate(&empty1);
  hipEventCreate(&empty2);
  hipEventCreate(&full1);
  hipEventCreate(&full2);            
    
  hipEventRecord(empty1, workStream);    
  hipEventRecord(empty2, workStream);

    
  for (;;) {
    hipEventSynchronize(empty1);
    hipMemcpyAsync(dBuff1, hBuff1, size, hipMemcpyHostToDevice, transferStream);
    hipEventRecord(full1, transferStream);
      
    hipEventSynchronize(empty2);
    hipMemcpyAsync(dBuff2, hBuff2, size, hipMemcpyHostToDevice, transferStream);
    hipEventRecord(full2, transferStream);
      
    hipEventSynchronize(full1);
    kernel<<<numBlocks, numThreads, numSharedBytes, workStream>>>(dBuff1);
    hipEventRecord(empty1, workStream);
      
    hipEventSynchronize(full2);
    kernel<<<numBlocks, numThreads, numSharedBytes, workStream>>>(dBuff2);
    hipEventRecord(empty2, workStream);

    /*
      hipEventElapsedTime(&time, empty2, full2);
      printf("Data transfer took: %f ms\n", time);
    */
  }
    
  hipStreamDestroy(transferStream);
  hipStreamDestroy(workStream);
    
  hipEventDestroy(empty1);
  hipEventDestroy(empty2);
  hipEventDestroy(full1);
  hipEventDestroy(full2);            
}