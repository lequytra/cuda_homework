
void addArray(const ELEMENT *dArray, INDEX size,
	      ELEMENT *dSum);

ELEMENT multiDeviceAddArray(const ELEMENT *hA,  INDEX size)
{
  
  int numDevices;
  CU(hipGetDeviceCount(&numDevices));
  
  INDEX numElementsPerDevice = size/numDevices;  // assume divisible
  
  // Input and output buffer for ech device and CPU
  ELEMENT **dA   = (ELEMENT **) malloc(numDevices*sizeof(ELEMENT*));
  ELEMENT **dSum = (ELEMENT **) malloc(numDevices*sizeof(ELEMENT*));
  ELEMENT  *hSum = (ELEMENT  *) malloc(numDevices*sizeof(ELEMENT));
  
  // Let each device work on its portion
  for (int dev = 0; dev < numDevices; dev++) {
    
    CU(hipSetDevice(dev));
    
    CU(hipMalloc((void**) &dA[dev],   sizeof(ELEMENT)*numElementsPerDevice));
    CU(hipMalloc((void**) &dSum[dev], sizeof(ELEMENT)));
    
    CU(hipMemcpyAsync(&dA[dev], hBuff + dev*numElementsPerDevice, hipMemcpyHostToDevice));
    
    addArray(dA[dev], numElementsPerDevice, dSum[dev]);   // async, independent of device
  }

  
  ////////// VERSION 1
  ////////// Get sums in order and add them up on CPU
  
  for (int dev = 0; dev < numDevices; dev++) {
    
    CU(hipSetDevice(dev));
    
    CU(hipMemcpyAsync(&hSum[dev], dSum[dev], sizeof(ELEMENT)));
  }
  
  ELEMENT sum = 0;
  
  for (int dev = 0; dev < numDevices; dev++) {
    
    CU(hipSetDevice(dev));
    
    CU(hipDeviceSynchronize());
    
    sum += hSum[dev];
  }
  
  return sum;
  
  
  
  

  ////////// VERSION 2
  ////////// Get sums in order and add them up on device 0
  
  
  
  
  // Enable peer access to device 0
  CU(hipSetDevice(0));
  
  for (int dev = 1; dev < numDevices; dev++) {
    
    CU(hipDeviceCanAccessPeer(&canAccess, 0, dev));  // 0 can access dev's memory? 
    
    if (canAccess) {
      CU(hipDeviceEnablePeerAccess(dev, 0));         // make dev accessible by current device
    } else {
      printf("Device 0 cannot access device %d\n", dev);
    }
  }
  
  
  // Transfer results from other devices to array dSums0 on device 0
  ELEMENT *dSums0;   
  CU(hipMalloc((void**) &dSums0, numDevices*sizeof(ELEMENT)));
  
  CU(hipMemcpyAsync(dSums0[0], dSum[0], hipMemcpyDeviceToDevice));        // resukt from device 0
  
  for (int dev = 1; dev < numDevices; dev++) {
    
    CU(hipMemcpyPeer(&dSums0[dev], 0, dSum[dev], dev, sizeof(ELEMENT)));   // results from other devices
  }
  
  addArray(dSums0, numDevices, dSum[0]);                                    // add upp all the results
  
  CU(hipMemcpy(&hSum, dSum[0], hipMemcpyDeviceToHost));
  
  return hSum;
  
  
  
  
  ////////// VERSION 3
  ////////// Get sums as they become available add them up on CPU
  
  
  hipEvent_t *processed = (hipEvent_t *) malloc(numDevices*sizeof(hipEvent_t));
  
  
  for (int dev = 0; dev < numDevices; dev++) {
    
    CU(hipEventCreate(&processed[dev]));
    
    CU(hipSetDevice(dev));
    
    CU(hipEventRecord(processed[dev]));
  }
  
  sum = 0;
  int numProcessed = 0;
  
  while (numProcessed < numDevices) {
    
    for (int dev = 0; dev < numDevices; dev++) {
      
      if (processed[dev] == NULL)                              continue; // already processed
      if (hipEventQuery(processed[dev]) == hipErrorNotReady) continue; // not ready
      
      CU(hipSetDevice(dev));
      
      CU(hipMemcpy(&hSum, dSum[dev], hipMemcpyDeviceToHost));         // get the result to CPU
      
      sum += hSum;                                                      // process the result
      
      processed[dev] = NULL;                                            // indicate that dev processed
      
      numProcessed++;                                                   // count how many processed
    }
  }
  
  return sum;
}
