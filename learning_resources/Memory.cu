
__host__ __device__ hipError_t hipFree ( void* devPtr );
__host__            hipError_t hipHostFree ( void* ptr );
__host__ __device__ hipError_t hipMalloc ( void** devPtr, size_t size );
__host__            hipError_t hipHostMalloc ( void** ptr, size_t size );
__host__            hipError_t hipMallocManaged ( void** devPtr, size_t size, unsigned int flags = hipMemAttachGlobal);
__host__            hipError_t hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind );
__host__ __device__ hipError_t hipMemcpyAsync ( void* dst, const void* src, size_t count, hipMemcpyKind kind, hipStream_t stream = 0 );
__host__            hipError_t hipMemset ( void* devPtr, int  value, size_t count );
__host__ __device__ hipError_t hipMemsetAsync ( void* devPtr, int  value, size_t count, hipStream_t stream = 0 );


void f()
{
  int size = 100;
  int numBlocks, numThreads;
  
  {// Basic allocation hipMalloc

    int *hBuff = (int *) malloc(size);
    hBuff[0] = 42;
  
    int *dBuff;
    hipMalloc(&dBuff, size);
    hipMemcpy(dBuff, hBuff, size, hipMemcpyHostToDevice);

    kernel<<<numBlocks, numThreads>>>(dBuff);
    
    hipFree(dBuff);
  }

  {// Allocation in pinned memory hipHostMalloc
    
    int *hBuff = (int *) malloc(size);
    hBuff[0] = 42;
    
    int *dBuff;
    hipHostMalloc(&dBuff, size);
    hipMemcpy(dBuff, hBuff, size, hipMemcpyHostToDevice);
    
    kernel<<<numBlocks, numThreads>>>(dBuff);
  
    hipHostFree(dBuff);
  }

    
  {// Unified Memory Model
    
    int *dBuff;
    hipMallocManaged(&dBuff, size);
    dBuff[0] = 42;
    
    kernel<<<numBlocks, numThreads>>>(dBuff);
    
    hipFree(dBuff);
  }
}