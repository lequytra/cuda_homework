

void CPUwaits()
{
  int numBlocks, numThreads, numSharedBytes, size;
  
  int *hBuff1, *hBuff2, *dBuff1, *dBuff2;

  float time;
    
  hipStream_t transferStream, workStream;

  CU(hipStreamCreate(&transferStream));
  CU(hipStreamCreate(&workStream));
    
  hipEvent_t empty1, empty2, full1, full2;

  CU(hipEventCreate(&empty1));
  CU(hipEventCreate(&empty2));
  CU(hipEventCreate(&full1));
  CU(hipEventCreate(&full2));            
    
  CU(hipEventRecord(empty1, workStream));    
  CU(hipEventRecord(empty2, workStream));

    
  for (;;) {
    CU(hipEventSynchronize(empty1));
    // wait for hBuff1 to be filled
    CU(hipMemcpyAsync(dBuff1, hBuff1, size, hipMemcpyHostToDevice, transferStream));
    CU(hipEventRecord(full1, transferStream));
      
    CU(hipEventSynchronize(empty2));
    // wait for hBuff2 to be filled
    CU(hipMemcpyAsync(dBuff2, hBuff2, size, hipMemcpyHostToDevice, transferStream));
    CU(hipEventRecord(full2, transferStream));
      
    CU(hipEventSynchronize(full1));
    kernel<<<numBlocks, numThreads, numSharedBytes, workStream>>>(dBuff1);
    CU(hipEventRecord(empty1, workStream));
      
    CU(hipEventSynchronize(full2));
    kernel<<<numBlocks, numThreads, numSharedBytes, workStream>>>(dBuff2);
    CU(hipEventRecord(empty2, workStream));

    /*
      CU(hipEventElapsedTime(&time, empty2, full2));
      printf("Data transfer took: %f ms\n", time);
    */
  }
    
  CU(hipStreamDestroy(transferStream));
  CU(hipStreamDestroy(workStream));
    
  CU(hipEventDestroy(empty1));
  CU(hipEventDestroy(empty2));
  CU(hipEventDestroy(full1));
  CU(hipEventDestroy(full2));            
}







void GPUwaits()
{
  int numBlocks, numThreads, numSharedBytes, size;
  
  int *hBuff1, *hBuff2, *dBuff1, *dBuff2;

  float time;
    
  hipStream_t transferStream, workStream;

  CU(hipStreamCreate(&transferStream));
  CU(hipStreamCreate(&workStream));
    
  hipEvent_t empty1, empty2, full1, full2;

  CU(hipEventCreate(&empty1));
  CU(hipEventCreate(&empty2));
  CU(hipEventCreate(&full1));
  CU(hipEventCreate(&full2));            
    
  CU(hipEventRecord(empty1, workStream));    
  CU(hipEventRecord(empty2, workStream));


  // host data already ready
  for (int i = 0; i < numPreparedData; i++) {
    
    CU(hipStreamWaitEvent(transferStream, empty1));
    CU(hipMemcpyAsync(dBuff1, hBuff1[i], size, hipMemcpyHostToDevice, transferStream));
    CU(hipEventRecord(full1, transferStream));
      
    CU(hipStreamWaitEvent(transferStream, empty2));
    CU(hipMemcpyAsync(dBuff2, hBuff2[i], size, hipMemcpyHostToDevice, transferStream));
    CU(hipEventRecord(full2, transferStream));
      
    CU(hipStreamWaitEvent(workStream, full1));
    kernel<<<numBlocks, numThreads, numSharedBytes, workStream>>>(dBuff1);
    CU(hipEventRecord(empty1, workStream));
      
    CU(hipStreamWaitEvent(workStream, full2));
    kernel<<<numBlocks, numThreads, numSharedBytes, workStream>>>(dBuff2);
    CU(hipEventRecord(empty2, workStream));

  }
    
  CU(hipStreamDestroy(transferStream));
  CU(hipStreamDestroy(workStream));
    
  CU(hipEventDestroy(empty1));
  CU(hipEventDestroy(empty2));
  CU(hipEventDestroy(full1));
  CU(hipEventDestroy(full2));            
}
