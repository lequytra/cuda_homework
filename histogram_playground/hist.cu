#include "hip/hip_runtime.h"
/*
 * Compile: nvcc hist.cu -o hist
 * Run: ./hist [options]
 * Options:
 *   --random       : Use random input values (default)
 *   --incrementing : Use incrementing input values
 *   --size N       : Set input array size (default: 1024)
 *   --bins N       : Set number of histogram bins (default: 256)
 * 
 * Example: ./hist --incrementing --size 2048 --bins 128
 */

#include <stdio.h>
#include <limits.h>
#include <stdlib.h>
#include <tuple>
#include <hip/hip_runtime.h>
#include <string.h>
#include "../CU.h"

typedef unsigned int    ELEMENT;
typedef long            INDEX; 

// Constants for ELEMENT type bounds
const ELEMENT ELEMENT_MAX = UINT_MAX;
const ELEMENT ELEMENT_MIN = 0;

__global__ void minmax(ELEMENT* input, ELEMENT* minOutput, ELEMENT* maxOutput, INDEX size) {
    INDEX startIdx = blockIdx.x * blockDim.x + threadIdx.x;
    INDEX t = threadIdx.x; 
    INDEX halfSize = blockDim.x / 2; 

    // copy data to shared memory, each thread handles 2
    extern __shared__ ELEMENT minIntermediate[]; 
    extern __shared__ ELEMENT maxIntermediate[]; 
    if (startIdx < size) {
        minIntermediate[t] = input[startIdx];
        maxIntermediate[t] = input[startIdx];
    }
    else {
        minIntermediate[t] = ELEMENT_MAX; 
        maxIntermediate[t] = ELEMENT_MIN; 
    }
    if (startIdx + halfSize < size) {
        minIntermediate[t + halfSize] = input[startIdx + halfSize];
        maxIntermediate[t + halfSize] = input[startIdx + halfSize];
    }
    else {
        minIntermediate[t] = ELEMENT_MAX; 
        maxIntermediate[t] = ELEMENT_MIN; 
    }

    // __syncthreads(); 
    // if (threadIdx.x == 0) {
    //     printf("Block %d intermediates: ", blockIdx.x);
    //     for (int i = 0; i < blockDim.x; i++) {
    //         printf("[%u,%u] ", minIntermediate[i], maxIntermediate[i]);
    //     }
    //     printf("\n");
    // }
    
    t <<= 1; 
    for (INDEX stride = 1; stride < halfSize; stride <<= 1) {
        __syncthreads(); 
        if (t % stride == 0) {
            minIntermediate[t] = min(minIntermediate[t], minIntermediate[t + stride]);
            maxIntermediate[t] = max(maxIntermediate[t], maxIntermediate[t + stride]);
        }
    }

    __syncthreads(); // Q: Do we need __synthreads here?
    if (threadIdx.x == 0) {
        // printf("Block %d intermediates: ", blockIdx.x);
        // for (int i = 0; i < blockDim.x; i++) {
        //     printf("[%u,%u] ", minIntermediate[i], maxIntermediate[i]);
        // }
        // printf("\n");
        minOutput[blockIdx.x] = minIntermediate[0];
        maxOutput[blockIdx.x] = maxIntermediate[0];
    }
}

void printUsage(const char* program_name) {
    printf("Usage: %s [options]\n", program_name);
    printf("Options:\n");
    printf("  --random       : Use random input values (default)\n");
    printf("  --incrementing : Use incrementing input values\n");
    printf("  --size N       : Set input array size (default: 1024)\n");
    printf("  --bins N       : Set number of histogram bins (default: 256)\n");
    printf("\nExample: %s --incrementing --size 2048 --bins 128\n", program_name);
}

int parseArguments(int argc, char** argv, bool* use_random, INDEX* size, INDEX* num_bins) {
    // Default parameters
    *use_random = true;
    *size = 1024;
    *num_bins = 256;
    
    // Parse command line arguments
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--incrementing") == 0) {
            *use_random = false;
        } else if (strcmp(argv[i], "--random") == 0) {
            *use_random = true;
        } else if (strcmp(argv[i], "--size") == 0) {
            if (i + 1 < argc) {
                *size = atoi(argv[++i]);
                if (*size <= 0) {
                    printf("Error: Size must be positive\n");
                    return 1;
                }
            } else {
                printf("Error: --size requires a value\n");
                return 1;
            }
        } else if (strcmp(argv[i], "--bins") == 0) {
            if (i + 1 < argc) {
                *num_bins = atoi(argv[++i]);
                if (*num_bins <= 0) {
                    printf("Error: Number of bins must be positive\n");
                    return 1;
                }
            } else {
                printf("Error: --bins requires a value\n");
                return 1;
            }
        } else {
            printf("Error: Unknown option '%s'\n", argv[i]);
            return 1;
        }
    }
    return 0;
}

void printArray(const char* name, const ELEMENT* array, INDEX size, INDEX max_print = 10) {
    printf("%s: [", name);
    for (int i = 0; i < size && i < max_print; i++) {
        printf("%u", array[i]);
        if (i < size - 1 && i < max_print - 1) {
            printf(", ");
        }
    }
    if (size > max_print) {
        printf(", ...");
    }
    printf("]\n");
}

INDEX powerOf2(INDEX n) {
    INDEX N; 
    for (N = 1; N < n; N <<= 1) {}
    return N; 
}

INDEX getMaxElements() {
    hipDeviceProp_t p; 
    hipGetDeviceProperties(&p, 0); 

    INDEX numElements = min(
        // we initialize 2 arrays in shared memory
        (int)(p.sharedMemPerBlock / (2 * sizeof(ELEMENT))), 
        2 * min(
            p.maxThreadsPerBlock,
            p.maxThreadsDim[0] // 1 thread handles 2 elments
        )
    );
    return powerOf2(numElements); 
}

void verifyMinMaxResults(ELEMENT gpuMin, ELEMENT gpuMax, ELEMENT cpuMin, ELEMENT cpuMax) {
    printf("\nVerifying results:\n");
    printf("GPU Min: %u, Max: %u\n", gpuMin, gpuMax);
    printf("CPU Min: %u, Max: %u\n", cpuMin, cpuMax);
    
    if (gpuMin == cpuMin && gpuMax == cpuMax) {
        printf("Results match! Verification successful.\n");
    } else {
        printf("ERROR: Results do not match!\n");
        if (gpuMin != cpuMin) printf("Min values differ - GPU: %u, CPU: %u\n", gpuMin, cpuMin);
        if (gpuMax != cpuMax) printf("Max values differ - GPU: %u, CPU: %u\n", gpuMax, cpuMax);
    }
}

int main(int argc, char** argv) {
    // Parameters to be set by parseArguments
    bool use_random;
    INDEX size;
    INDEX num_bins;
    
    // Parse command line arguments
    if (parseArguments(argc, argv, &use_random, &size, &num_bins) != 0) {
        printUsage(argv[0]);
        return 1;
    }

    INDEX sizePadded = powerOf2(size); 
    
    printf("Running with parameters:\n");
    printf("  Input type: %s\n", use_random ? "random" : "incrementing");
    printf("  Array size: %ld, padded: %ld\n", size, sizePadded);
    printf("  Number of bins: %ld\n", num_bins);
    
    // Host memory allocation
    ELEMENT* hInput = (ELEMENT*)malloc(size * sizeof(ELEMENT));
    ELEMENT* h_output = (ELEMENT*)malloc(num_bins * sizeof(ELEMENT));

    
    // Initialize input array
    if (use_random) {
        printf("Using random input values\n");
        for (int i = 0; i < size; i++) {
            hInput[i] = rand();  // Values between 0 and num_bins-1
        }
    } else {
        printf("Using incrementing input values\n");
        for (int i = 0; i < size; i++) {
            hInput[i] = i;  // Values cycle from 0 to num_bins-1
        }
    }
    // pad zero
    for (INDEX i = size; i < sizePadded; i++) {
        hInput[i] = 0; 
    }
    
    // Print first few elements of input array
    printArray("Input array", hInput, size);
    
    // Device memory allocation
    ELEMENT* dInput;
    CU(hipMalloc((void**)&dInput, size * sizeof(ELEMENT)));
    // Copy input data to device
    CU(hipMemcpy(dInput, hInput, size * sizeof(ELEMENT), hipMemcpyHostToDevice));
    
    INDEX maxElements = min(getMaxElements(), sizePadded);
    INDEX numElements = sizePadded; 
    int numBlocks = ceil(numElements / maxElements); 
    int numThreads = min(numElements, maxElements); 

    ELEMENT* dOutputMin;
    ELEMENT* dOutputMax;
    
    CU(hipMalloc((void**)&dOutputMin, numBlocks * sizeof(ELEMENT)));
    CU(hipMalloc((void**)&dOutputMax, numBlocks * sizeof(ELEMENT)));
    INDEX curSize = size; 
    ELEMENT hMin, hMax;
    // Launch kernel
    for (;;) {
        numBlocks = ceil(numElements / maxElements);

        numThreads = maxElements / 2; 
        printf("  Blocks: %d, Threads: %d\n", numBlocks, numThreads);

        minmax<<<numBlocks, numThreads, maxElements * 2 * sizeof(ELEMENT)>>>(dInput, dOutputMin, dOutputMax, curSize); 

        numElements = numBlocks; 
        curSize = numElements; 

        if (numElements == 1) {
            CU(hipDeviceSynchronize());
            CU(hipMemcpy(&hMin, dOutputMin, sizeof(ELEMENT), hipMemcpyDeviceToHost));
            CU(hipMemcpy(&hMax, dOutputMax, sizeof(ELEMENT), hipMemcpyDeviceToHost));
            break;
        }
    }
    
    // CPU verification
    ELEMENT cpuMin = ELEMENT_MAX;
    ELEMENT cpuMax = ELEMENT_MIN;
    
    // Simple sequential min/max calculation for verification
    for (INDEX i = 0; i < size; i++) {
        cpuMin = min(cpuMin, hInput[i]);
        cpuMax = max(cpuMax, hInput[i]);
    }
    
    // Compare GPU and CPU results
    verifyMinMaxResults(hMin, hMax, cpuMin, cpuMax);
    
    // Free device memory
    CU(hipFree(dInput));
    CU(hipFree(dOutputMin));
    CU(hipFree(dOutputMax));
    
    // Free host memory
    free(hInput);
    
    printf("Program completed successfully!\n");
    return 0;
}
