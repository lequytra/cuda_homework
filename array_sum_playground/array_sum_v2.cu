#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>

// nvcc -o array_sum_v2 array_sum_v2.cu && ./array_sum_v2

const int MAX_THREADS_PER_BLOCK = 1024;


__global__ void computeSum(
    float* input, float* output, int size
) {
    int defaultHalfSize = blockDim.x; 
    int defaultSegmentSize = blockDim.x * 2;
    // the last block may have smaller segment
    bool sizeDivisible = size % defaultSegmentSize == 0;
    // last block except when we only launch 1 block
    bool isLastBlock = (gridDim.x - 1 == blockIdx.x) * blockIdx.x;
    int curHalfSize = 
        isLastBlock * (ceil((size % defaultSegmentSize) / 2.0f)) + // when the last block has smaller segment
        isLastBlock * sizeDivisible * defaultHalfSize +
        !isLastBlock * defaultHalfSize;
    
    int inputIdx = threadIdx.x + blockIdx.x * defaultHalfSize * 2;
    int outputIdx = threadIdx.x + blockIdx.x * defaultHalfSize; 

    // printf("Thread %d Block %d CurHalfSize %d GridDim %d BlockDim %d\n  size %% blockDim.x = %d\n  ceil((size %% blockDim.x) / 2.0f) = %f\n",
    //        threadIdx.x, blockIdx.x, curHalfSize, gridDim.x, blockDim.x,
    //        size % blockDim.x,
    //        ceil((size % blockDim.x) / 2.0f));

    if (threadIdx.x < curHalfSize) {
        if (inputIdx + curHalfSize < size) {
            output[outputIdx] = input[inputIdx] + input[inputIdx + curHalfSize]; 
        }
        else {
            output[outputIdx] = input[inputIdx];
        }
    }
}

int main() {
    srand(time(NULL)); // Set random seed based on current time

    int N = rand() % (1024 * 8) + 10;
    float h_A[N];
    // Initialize array with random values
    for (int i = 0; i < N; i++) {
        h_A[i] = (float)rand() / RAND_MAX;
    }

    float* d_A;
    hipMalloc((void**) &d_A, N * sizeof(float));
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);

    int halfSize = ceil(N / 2.0); 

    float* d_B;
    hipMalloc((void**) &d_B, halfSize * sizeof(float));
    hipMemset(d_B, 0, halfSize * sizeof(float));

    int curSize = N; 
    int numBlocks = 0;
    int numThreads = 0; 

    bool isAInput = true;
    float *inputPtr, *outputPtr; 

    while (curSize > 1) {
        numBlocks = ceil(halfSize / float(MAX_THREADS_PER_BLOCK));
        numThreads = min(halfSize, MAX_THREADS_PER_BLOCK);

        if (isAInput) {
            inputPtr = d_A; 
            outputPtr = d_B;
        } else {
            inputPtr = d_B; 
            outputPtr = d_A; 
        }

        computeSum<<<numBlocks, numThreads>>>(inputPtr, outputPtr, curSize);

        curSize = halfSize; 
        halfSize = ceil(curSize / 2.0);
        isAInput = !isAInput;
    }

    hipDeviceSynchronize();

    float result;
    hipMemcpy(&result, outputPtr, sizeof(float), hipMemcpyDeviceToHost);

    // printf("Input array: ");
    // for (int i = 0; i < N; i++) {
    //     printf("%.2f ", h_A[i]);
    // }
    printf("\nN: %.2d\n", N);
    printf("\nSum: %.2f\n", result);

    // Calculate expected sum on CPU for verification
    float expected_sum = 0.0f;
    for (int i = 0; i < N; i++) {
        expected_sum += h_A[i];
    }
    printf("Expected sum: %.2f\n", expected_sum);
    printf("Difference: %.2f\n", fabs(expected_sum - result));

    hipFree(d_A);

}

